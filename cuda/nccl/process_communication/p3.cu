
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#define DSIZE 1
int main(){
	int * data;
	hipIpcMemHandle_t data_handle;
	char handle_buffer[sizeof(data_handle)+1];
	memset(handle_buffer, 0, sizeof(data_handle)+1);
	FILE * fp;
	fp = fopen("handlepipe2", "r");
	for (int i = 0; i < sizeof(data_handle); i++){
		fscanf(fp,"%c", handle_buffer+i);
	}
	memcpy((char *)(&data_handle), handle_buffer, sizeof(data_handle));
	hipIpcOpenMemHandle((void **)&data, data_handle, hipIpcMemLazyEnablePeerAccess);
	int * result = (int *) malloc(DSIZE*sizeof(int));
	hipMemcpy(result, data, sizeof(int), hipMemcpyDeviceToHost);
	printf("result: %d\n", *result);
	return 0;
}
