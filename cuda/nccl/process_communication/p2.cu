
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>

int main(){
	int * data;
	int i = 12;
	hipIpcMemHandle_t data_handle;
	char handle_buffer[sizeof(data_handle)+1];
	memset(handle_buffer, 0, sizeof(data_handle)+1);
	FILE * fp;
	fp = fopen("handlepipe", "r");
	for (int i = 0; i < sizeof(data_handle); i++){
		fscanf(fp,"%c", handle_buffer+i);
	}
	memcpy((char *)(&data_handle), handle_buffer, sizeof(data_handle));
	hipIpcOpenMemHandle((void **)&data, data_handle, hipIpcMemLazyEnablePeerAccess);
	hipMemcpy(data, &i, sizeof(int), hipMemcpyHostToDevice);
	printf("changed value\n");
	return 0;
}
