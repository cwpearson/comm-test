
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

int main(int argc, char const **argv)
{
	int nStreams = 4;
	int blockSize = 256;
	int n = 4 * 1024 * blockSize * nStreams;
	int streamSize = n / nStreams;
	int streamBytes = streamSize * sizeof(float);
	int bytes = n * sizeof(float);

	int devId = 0;
	if (argc > 1) devId = atoi(argv[1]);

	hipDeviceProp_t prop;
	checkCuda( hipGetDeviceProperties(&prop, devId));
	printf("Device : %s\n", prop.name);
	checkCuda( hipSetDevice(devId) );

	// allocate pinned host memory and device memory
	// non unified
	float *a, *d_a;
	checkCuda( hipHostMalloc((void**)&a, bytes, hipHostMallocDefault) );      // host pinned
	checkCuda( hipMalloc((void**)&d_a, bytes) ); // device

	float ms; // elapsed time in milliseconds

	//create streams
	hipStream_t stream[nStreams];

	//events for profiling
	hipEvent_t startEvent, stopEvent; 

	checkCuda( hipEventCreate(&startEvent) );
	checkCuda( hipEventCreate(&stopEvent) );
	for (int i = 0; i < nStreams; ++i) checkCuda( hipStreamCreate(&stream[i]) );

	//sequential memory transfer 
	memset(a, 0, bytes);
	checkCuda( hipEventRecord(startEvent) );
	checkCuda( hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice) );
	checkCuda( hipEventRecord(stopEvent) );
	checkCuda( hipEventSynchronize(stopEvent));
	checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
	printf(" Sequential Memory Test\n");
	printf("  Host to Device bandwidth (GB/s): %f\n", bytes * 1e-6 / ms);
	printf(" Host to device tranfser time (ms): %f\n", ms);
	
	checkCuda( hipEventRecord(startEvent));
	checkCuda( hipMemcpy(a, d_a, bytes, hipMemcpyDeviceToHost) );
	checkCuda( hipEventRecord(stopEvent));
	checkCuda( hipEventSynchronize(stopEvent) );
	checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
	printf("  Device to Host bandwidth (GB/s): %f\n", bytes * 1e-6 / ms);
	printf(" Device to host tranfser time (ms): %f\n", ms);

	//asynchronous version: loop  over streams{copy H2D, copy D2H}
	memset(a, 0, bytes);
	checkCuda( hipEventRecord(startEvent,0) );
	for (int i = 0; i < nStreams; ++i) {
		int offset = i * streamSize;
		checkCuda( hipMemcpyAsync(&d_a[offset], &a[offset], 
		                           streamBytes, hipMemcpyHostToDevice, 
		                           stream[i]) );
	}
	checkCuda( hipEventRecord(stopEvent, 0) );
	checkCuda( hipEventSynchronize(stopEvent) );
	checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
	printf( "Asynchronous Memory Test\n");
	printf("  Host to Device bandwidth (GB/s): %f\n", bytes * 1e-6 / ms);
	printf(" Host to device tranfser time (ms): %f\n", ms);

	checkCuda( hipEventRecord(startEvent,0) );
	for (int i = 0; i < nStreams; ++i) {
		int offset = i*streamSize;
		checkCuda( hipMemcpyAsync(&a[offset], &d_a[offset], 
		                           streamBytes, hipMemcpyDeviceToHost,
		                           stream[i]) );
	}
	checkCuda( hipEventRecord(stopEvent));
	checkCuda( hipEventSynchronize(stopEvent) );
	checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
	printf("  Device to Host bandwidth (GB/s): %f\n", bytes * 1e-6 / ms);
	printf(" Device to host tranfser time (ms): %f\n", ms);


	// cleanup
	checkCuda( hipEventDestroy(startEvent) );
	checkCuda( hipEventDestroy(stopEvent) );
	for (int i = 0; i < nStreams; ++i)
		checkCuda( hipStreamDestroy(stream[i]) );
	hipFree(d_a);
	hipHostFree(a);

	return 0;
}
