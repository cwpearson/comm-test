#include <nccl.h>

typedef struct {
  double* sendBuff;
  double* recvBuff;
  int size;
  hipStream_t stream;
} PerThreadData;

int main(int argc, char* argv[])
{
  int nGPUs;
  hipGetDeviceCount(&nGPUs);
  ncclComm_t* comms = (ncclComm_t*)malloc(sizeof(ncclComm_t)*nGPUs);
  ncclCommInitAll(comms, nGPUs); // initialize communicator
                                // One communicator per process

  PerThreadData* data;

  ... // Allocate data and issue work to each GPU's
      // perDevStream to populate the sendBuffs.

  for(int i=0; i<nGPUs; ++i) {
    hipSetDevice(i); // Correct device must be set
                      // prior to each collective call.
    ncclAllReduce(data[i].sendBuff, data[i].recvBuff, size,
        ncclDouble, ncclSum, comms[i], data[i].stream);
  }

  ... // Issue work into data[*].stream to consume buffers, etc.
}
