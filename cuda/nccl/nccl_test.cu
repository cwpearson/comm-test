
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#include <nccl.h>

#define CUDA_MUST(ans) \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

#define NCCL_MUST(ans) \
{ ncclAssert((ans), __FILE__, __LINE__); }
inline void ncclAssert(ncclResult_t result, const char *file, int line,
                      bool abort = true) {
  if (result != ncclSuccess) {
    fprintf(stderr, "nnclAssert: %s %s %d\n", ncclGetErrorString(result), file,
            line);
    if (abort)
      exit(result);
  }
}



typedef struct {
  double* send_;
  double* recv_;
  hipStream_t stream_;
} GPUBuffer;

__global__ void kernel() {}

int main(int argc, char* argv[]) {
  constexpr int BUFFER_SIZE = 1024;

  int nGPUs;
  CUDA_MUST(hipGetDeviceCount(&nGPUs));

  if (nGPUs == 0) {
	  std::cerr << "No devices found!\n";
	  exit(EXIT_FAILURE);
  } else {
	std::cout << nGPUs << " devices!" << std::endl;
  }

  // Data that will be on each gpu
  std::vector<std::vector<double>> test_data(nGPUs, std::vector<double>(BUFFER_SIZE, 1.0));
  std::vector<double> test_results(nGPUs);
  std::cout << "Allocated host data" << std::endl;

  // associate all devices with rank 0
  std::vector<ncclComm_t> comms(nGPUs);
  std::vector<int> devList(nGPUs, 0); 
  NCCL_MUST(ncclCommInitAll(&comms[0], nGPUs, &devList[0]));
  std::cout << "Initialized communicators." << std::endl;


  // Create buffers on GPUs
  std::vector<GPUBuffer> GPUBuffers(nGPUs);
  for (auto dev = 0; dev < GPUBuffers.size(); ++dev) {
	  auto &buf = GPUBuffers[dev];
	  CUDA_MUST(hipStreamCreate(&buf.stream_));
	  CUDA_MUST(hipSetDevice(dev));
	  CUDA_MUST(hipMalloc(&buf.send_, BUFFER_SIZE));
	  CUDA_MUST(hipMalloc(&buf.recv_, 1)); // one value for all-reduce
  }
  std::cout << "Created buffers" << std::endl;

  // Copy test data to GPUs
  for (auto i = 0; i < GPUBuffers.size(); ++i) {
    auto &buf = GPUBuffers[i];
    fprintf(stderr, "%d) %p <--(%lu)-- %p\n", i, buf.send_, test_data[i].size(), &test_data[i][0]);
    CUDA_MUST(hipSetDevice(i));
    CUDA_MUST(hipMemcpy(buf.send_, 
			 &test_data[i][0], 
			 sizeof(double)*test_data[i].size(), 
			 hipMemcpyHostToDevice)
	     );
  }
  std::cout << "Copied H2D" << std::endl;

  for(int i=0; i<nGPUs; ++i) {
    hipSetDevice(i); // Correct device must be set
                      // prior to each collective call.
    auto &buf = GPUBuffers[i];
    ncclAllReduce(buf.send_, buf.recv_, BUFFER_SIZE,
        ncclDouble, ncclSum, comms[i], buf.stream_);
  }

  // Check results!
  for (auto i = 0; i < GPUBuffers.size(); ++i) {
    auto &buf = GPUBuffers[i];
    CUDA_MUST(hipSetDevice(i));
    CUDA_MUST(hipMemcpy(&test_results[i],
			 buf.recv_, 
			 sizeof(double), 
			 hipMemcpyDeviceToHost)
	     );
  }

  for (const auto &e : test_results) {
	  std::cout << e << " ";
  }
  std::cout <<std::endl;


  for (auto &buf : GPUBuffers) {
	  CUDA_MUST(hipStreamDestroy(buf.stream_));
	  CUDA_MUST(hipFree(buf.send_));
	  CUDA_MUST(hipFree(buf.recv_));
  }
}

